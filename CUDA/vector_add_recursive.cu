
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

double cpuTimer() {
  struct timeval clock;
  gettimeofday(&clock, NULL);
  
  return ((double) clock.tv_sec + (double) clock.tv_usec * 1.e-6);
}

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

  const int N = 2<<29;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;
  float *d;
  float *e;
  float *f;
  float *g;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);
  hipMallocManaged(&d, size);
  hipMallocManaged(&e, size);
  hipMallocManaged(&f, size);
  hipMallocManaged(&g, size);

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);
  initWith(0, d, N);
  initWith(5, e, N);
  initWith(0, f, N);
  initWith(6, g, N);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 256;
  numberOfBlocks = 32 * numberOfSMs;

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  double start = cpuTimer(), end;
  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));
  
  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(d, c, e, N);

  addVectorsErr = hipGetLastError();
  if (addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if (asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(f, d, g, N);
  
  addVectorsErr = hipGetLastError();
  if (addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if (asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  end = cpuTimer();

  double rtime = end - start;
  printf("Kernels executed in %lf seconds.\n", rtime);

  checkElementsAre((3 + 4 + 5 + 6), f, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}

